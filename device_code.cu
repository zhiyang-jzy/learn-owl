#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2019-2020 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "device_code.h"
#include <optix_device.h>

OPTIX_RAYGEN_PROGRAM(simpleRayGen)()
{
    const RayGenData &self = owl::getProgramData<RayGenData>();
    const vec2i pixelID = owl::getLaunchIndex();
    if (pixelID == owl::vec2i(0)) {
        printf("%sHello OptiX From your First RayGen Program%s\n",
               OWL_TERMINAL_CYAN,
               OWL_TERMINAL_DEFAULT);
    }

    vec3f light_pos(3,3,3);

    const vec2f screen = (vec2f(pixelID)+vec2f(.5f)) / vec2f(self.fbSize);
    owl::Ray ray;
    ray.origin
            = self.camera.pos;
    ray.direction
            = normalize(self.camera.dir_00
                        + screen.u * self.camera.dir_du
                        + screen.v * self.camera.dir_dv);

    RayData rayData;
    owl::traceRay(/*accel to trace against*/self.world,
            /*the ray to trace*/ray,
            /*prd*/rayData);

    vec3f color(1,1,1);

    const int fbOfs = pixelID.x+self.fbSize.x*pixelID.y;
    if(rayData.hit)
    {

        vec3f normal = normalize(rayData.normal);
        vec3f light_dir = normalize(light_pos-rayData.point);
        vec3f view_dir = normalize(ray.origin-rayData.point);
        vec3f halfway_dir = normalize(light_dir+view_dir);

        vec3f ambient = 0.05f * color;//环境光


        float diff = max(dot(light_dir,normal), 0.f);
        vec3f diffuse = diff * color; //漫反射

        float spec = pow(max(dot(normal,halfway_dir), 0.f), 128);
        vec3f specular = vec3f(0.3, 0.3, 0.3) * spec;// 镜面反射

        vec3f res = ambient + diffuse + specular;


        self.fbPtr[fbOfs]
                = owl::make_rgba(res);
    }
    else
        self.fbPtr[fbOfs] = owl::make_rgba(vec3f(0));
}

OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)()
{
    RayData &prd = owl::getPRD<RayData>();

    const TrianglesGeomData &self = owl::getProgramData<TrianglesGeomData>();

    // compute normal:
    const int   primID = optixGetPrimitiveIndex();
    const vec3i index  = self.index[primID];

//    const vec3f Ng     = normalize(cross(B-A,C-A));
    vec2f uv =  optixGetTriangleBarycentrics();
    vec3f normal
            = (1.f-uv.x-uv.y)*self.normal[index.x]
              +      uv.x      *self.normal[index.y]
              +           uv.y *self.normal[index.z];
    normal = normalize(normal);
    optixTransformNormalFromObjectToWorldSpace(normal);

    const vec3f org  = optixGetWorldRayOrigin();
    const vec3f dir  = optixGetWorldRayDirection();
    const float hit_t = optixGetRayTmax();
    const vec3f hit_P = org + hit_t * dir;
    prd.normal = normal;
    prd.point = hit_P;
    prd.hit = true;

}

OPTIX_MISS_PROGRAM(miss)()
{
    const vec2i pixelID = owl::getLaunchIndex();

    const MissProgData &self = owl::getProgramData<MissProgData>();

    RayData &prd = owl::getPRD<RayData>();
    int pattern = (pixelID.x / 8) ^ (pixelID.y/8);
    prd.hit = false;

}

